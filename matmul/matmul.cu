
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include <cstdlib>

inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void naiveMatmul(float *a, float *b, float *c, int M, int K, int N) {
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  int col = threadIdx.x + blockIdx.x * blockDim.x;
  float sum = 0.0f;
  for (int i = 0; i < K; ++i) {
    sum += a[row * K + i] *b[i * N + col];
  }
  c[row * N + col] = sum;
}

int main(int argc, char * argv[]) {
  // check command param
  if (argc < 4) {
    printf("Usage: ./matmul M K N\n");
    return 0;
  }

  int M = atoi(argv[1]);
  int K = atoi(argv[2]);
  int N = atoi(argv[3]);

  // malloc host memory
  float *a, *b, *c;
  a = (float*)malloc(M * K * sizeof(float));
  b = (float*)malloc(K * N * sizeof(float));
  c = (float*)malloc(M * N * sizeof(float));

  // initial data
  for (int i = 0; i < M * K; ++i) {
    a[i] = 1.0f;
  }
  for (int i = 0; i < K * N; ++i) {
    b[i] = 1.0f;
  }

  // malloc device memory
  float *d_a, *d_b, *d_c;
  hipMalloc((void**)&d_a, M * K * sizeof(float));
  hipMalloc((void**)&d_b, K * N * sizeof(float));
  hipMalloc((void**)&d_c, M * N * sizeof(float));

  // memcpy data from host to device
  hipMemcpy((void*)d_a, (void*)a, M * K * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy((void*)d_b, (void*)b, K * N * sizeof(float), hipMemcpyHostToDevice);

  // setup param of launch kernel
  dim3 blockSize(32, 32);
  dim3 gridSize((M + 31) / 32, (N + 31) / 32);

  // launch kernel
  // warm up
  naiveMatmul<<<gridSize, blockSize>>>(d_a, d_b, d_c, M, K, N);
  float ms;
  hipEvent_t startEvent, stopEvent;
  checkCuda(hipEventCreate(&startEvent));
  checkCuda(hipEventCreate(&stopEvent));
  checkCuda(hipEventRecord(startEvent,0));
  naiveMatmul<<<gridSize, blockSize>>>(d_a, d_b, d_c, M, K, N);
  checkCuda(hipEventRecord(stopEvent,0));
  checkCuda(hipEventSynchronize(stopEvent));
  checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));

  // memcpy result from device to host
  hipMemcpy((void*)c, (void*)d_c, M * N * sizeof(float), hipMemcpyDeviceToHost);

  // compare result
  float maxError = 0.0;
  for (int i = 0; i < M * N; i++)
    maxError = fmax(maxError, fabs(c[i] - K));
  std::cout << "Max Error: " << maxError << std::endl;

  // free device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  // free host memory
  free(a);
  free(b);
  free(c);

  return 0;
}
