
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>
#include <math.h>

#define N 800
#define BLOCK 256
inline hipError_t checkCuda(hipError_t result) {
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void naiveMatmul_2d(float *a, float *b, float *c, size_t pitch_a,
                               size_t pitch_b, size_t pitch_c) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int row = tid / N;
  int col = tid % N;
  float sum = 0.0f;
  if (row < N && col < N) {
    for (int i = 0; i < N; ++i) {
      sum += a[row * pitch_a / sizeof(float) + i] *
             b[i * pitch_b / sizeof(float) + col];
    }
    c[row * pitch_c / sizeof(float) + col] = sum;
  }
}

int main(int argc, char *argv[]) {
  printf("Usage: ./matmul\n");

  // malloc host memory
  float *a, *b, *c;
  a = (float *)malloc(N * N * sizeof(float));
  b = (float *)malloc(N * N * sizeof(float));
  c = (float *)malloc(N * N * sizeof(float));

  // initial data
  for (int i = 0; i < N * N; ++i) {
    a[i] = 1.0f;
    b[i] = 1.0f;
  }

  // malloc device memory
  float *d_a, *d_b, *d_c;
  size_t pitch_a, pitch_b, pitch_c;
  hipMallocPitch((void **)&d_a, &pitch_a, N * sizeof(float), N);
  hipMallocPitch((void **)&d_b, &pitch_b, N * sizeof(float), N);
  hipMallocPitch((void **)&d_c, &pitch_c, N * sizeof(float), N);

  // memcpy data from host to device
  hipMemcpy2D((void *)d_a, pitch_a, (void *)a, N * sizeof(float),
               N * sizeof(float), N, hipMemcpyHostToDevice);
  hipMemcpy2D((void *)d_b, pitch_b, (void *)b, N * sizeof(float),
               N * sizeof(float), N, hipMemcpyHostToDevice);

  // setup param of launch kernel
  dim3 blockSize(BLOCK);
  dim3 gridSize(N * (N + BLOCK - 1) / BLOCK);

  // launch kernel
  // warm up
  naiveMatmul_2d<<<gridSize, blockSize>>>(d_a, d_b, d_c, pitch_a, pitch_b,
                                          pitch_c);
  float ms;
  int repeat = 100;
  hipEvent_t startEvent, stopEvent;
  checkCuda(hipEventCreate(&startEvent));
  checkCuda(hipEventCreate(&stopEvent));
  checkCuda(hipEventRecord(startEvent, 0));
  for (int i = 0; i < repeat; ++i) {
    naiveMatmul_2d<<<gridSize, blockSize>>>(d_a, d_b, d_c, pitch_a, pitch_b,
                                            pitch_c);
  }
  checkCuda(hipEventRecord(stopEvent, 0));
  checkCuda(hipEventSynchronize(stopEvent));
  checkCuda(hipEventElapsedTime(&ms, startEvent, stopEvent));
  printf("time: %f ms\n", ms / repeat);

  // memcpy result from device to host
  hipMemcpy2D((void *)c, N * sizeof(float), (void *)d_c, pitch_c,
               N * sizeof(float), N, hipMemcpyDeviceToHost);

  // compare result
  float maxError = 0.0;
  for (int i = 0; i < N * N; i++)
    maxError = fmax(maxError, fabs(c[i] - N));
  std::cout << "Max Error: " << maxError << std::endl;

  // free device memory
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  // free host memory
  free(a);
  free(b);
  free(c);

  return 0;
}
